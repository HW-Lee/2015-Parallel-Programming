#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define OUTFILE 1
#define INF (int) 1 << 24

__device__ __host__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

int* Dist;
int* Dist_d;

__global__ void updateList(int* D, int blocksize, int N, int r, int blockDimWidth, int phase) {
    int bi, bj;
    switch(phase) {
        case 0:
            bi = r;
            bj = r;
            break;
        case 1:
            if (blockIdx.x == 1) {
                bj = r;
                bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            } else {
                bi = r;
                bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            }
            break;
        case 2:
            bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            bj = (r + blockIdx.x + 1) % (int) ceil((double) N/blocksize);
            break;
    }
    extern __shared__ int DS[];

    int offset_i = blocksize * bi;
    int offset_j = blocksize * bj;
    int offset_r = blocksize * r;

    // DS[0:bibs-1][:] = B[bi][bj] = D[bibs:(bi+1)bs-1][bjbs:(bj+1)bs-1]
    // DS[bibs:2bibs-1][:] = B[bi][r] = D[bibs:(bi+1)bs-1][rbs:(r+1)bs-1]
    // DS[2bibs:3bibs-1][:] = B[r][bi] = D[rbs:(r+1)bs-1][bjbs:(bj+1)bs-1]
    for (int i = threadIdx.y; i < blocksize && i+offset_i < N; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize && j+offset_j < N; j+=blockDimWidth) {
            DS[ij2ind(i, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_j, N)];
            DS[ij2ind(i+blocksize, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_r, N)];
            DS[ij2ind(i+2*blocksize, j, blocksize)] = D[ij2ind(i+offset_r, j+offset_j, N)];
        }
    }
    __syncthreads();

    // DS[i][j] = min{ DS[i][j], DS[i+bs][k] + DS[k+2bs][j] }
    for (int k = 0; k < blocksize; k++) {
        for (int i = threadIdx.y; i < blocksize && i+offset_i < N; i+=blockDimWidth) {
            for (int j = threadIdx.x; j < blocksize && j+offset_j < N; j+=blockDimWidth) {
                if (DS[ij2ind(i, j, blocksize)] > DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)]) {
                    DS[ij2ind(i, j, blocksize)] = DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)];
                    if (r == bi) DS[ij2ind(i+2*blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                    if (r == bj) DS[ij2ind(i+blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                }
            }
        }
        __syncthreads();
    }

    for (int i = threadIdx.y; i < blocksize && i+offset_i < N; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize && j+offset_j < N; j+=blockDimWidth) {
            // DS[i][j] = D[i+bsbi][j+bsbj]
            D[ij2ind(i+offset_i, j+offset_j, N)] = DS[ij2ind(i, j, blocksize)];
        }
    }
}

int main(int argc, char* argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, dev);

    // printf("dp.sharedMemPerBlock = %d\n", dp.sharedMemPerBlock);
    // printf("dp.maxThreadsPerBlock = %d\n", dp.maxThreadsPerBlock);
    // printf("dp.maxThreadsDim = (%d, %d, %d)\n", dp.maxThreadsDim[0], dp.maxThreadsDim[1], dp.maxThreadsDim[2]);
    // printf("dp.maxGridSize = (%d, %d, %d)\n", dp.maxGridSize[0], dp.maxGridSize[1], dp.maxGridSize[2]);

    int blockDimWidth = (int) sqrt(dp.maxThreadsPerBlock);
    dim3 block(blockDimWidth, blockDimWidth);

    if (argc < 3) {
        printf("not enough arguments.\n");
        return 0;
    }

    int blocksize;
    int MAX_BLOCKSIZE = (int) sqrt(dp.sharedMemPerBlock/3.0/sizeof(int));
    if (argc >= 4) blocksize = atoi(argv[3]);
    else blocksize = blockDimWidth;

    if (blocksize > MAX_BLOCKSIZE) blocksize = MAX_BLOCKSIZE;

    printf("Blocking factor: %d\n", blocksize);

    // TODO: Read file and get meta data
    FILE *infile = fopen(argv[1], "r");
    int m;
    int N;
    fscanf(infile, "%d %d", &N, &m);

    if (blocksize > N) blocksize = N;
    int N_ext = N + (blocksize - N % blocksize);

    // TODO: Allocate memory (pinned/unpinned)
    hipHostMalloc((void**) &Dist, sizeof(int) * N_ext*N_ext, hipHostMallocDefault);
    hipMalloc((void**) &Dist_d, sizeof(int) * N_ext*N_ext);

    for (int i = 0; i < N_ext; ++i) {
        for (int j = 0; j < N_ext; ++j) {
            if (i == j) Dist[ij2ind(i, j, N_ext)] = 0;
            else        Dist[ij2ind(i, j, N_ext)] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        Dist[ij2ind(a, b, N_ext)] = v;
    }

    // TODO: Copy values loaded from the file
    hipMemcpy((void*) Dist_d, (void*) Dist, sizeof(int) * N_ext*N_ext, hipMemcpyHostToDevice);

    // TODO: Updating list
    int num_blocks_per_column = (int) ceil((double) N_ext/blocksize);
    dim3 grid_1(2, num_blocks_per_column-1);
    dim3 grid_2(num_blocks_per_column-1, num_blocks_per_column-1);
    for (int r = 0; r < num_blocks_per_column; r++) {
        printf("\rCompute progress: %.2f%%", (float) r/num_blocks_per_column*100);
        updateList<<< 1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 0);
        updateList<<< grid_1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 1);
        updateList<<< grid_2, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 2);
        hipMemcpy((void*) Dist, (void*) Dist_d, sizeof(int), hipMemcpyDeviceToHost);
        printf("\rCompute progress: %.2f%%", (float) (r+1)/num_blocks_per_column*100);
    }
    printf("\n");

    // TODO: Copy final values
    hipMemcpy((void*) Dist, (void*) Dist_d, sizeof(int) * N_ext*N_ext, hipMemcpyDeviceToHost);

    // TODO: Write file
    if (OUTFILE == 1) {
        printf("Writing the file...\n");
        FILE *outfile = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (Dist[ij2ind(i, j, N_ext)] >= INF) fprintf(outfile, "INF ");
                else                                  fprintf(outfile, "%d ", Dist[ij2ind(i, j, N_ext)]);
            }
            fprintf(outfile, "\n");
        }
    }

    // TODO: Free memory
    hipHostFree(Dist);
    hipFree(Dist_d);

    return 0;
}
