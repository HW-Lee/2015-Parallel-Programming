#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define OUTFILE 1
#define INF (int) 1 << 24

__device__ __host__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

int* Dist;
int* Dist_d;

__global__ void updateList(int* D, int blocksize, int N, int r, int blockDimWidth, int phase) {
    int bi, bj;
    switch(phase) {
        case 0:
            bi = r;
            bj = r;
            break;
        case 1:
            if (blockIdx.x == 1) {
                bj = r;
                bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            } else {
                bi = r;
                bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            }
            break;
        case 2:
            bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            bj = (r + blockIdx.x + 1) % (int) ceil((double) N/blocksize);
            break;
    }
    extern __shared__ int DS[];

    int offset_i = blocksize * bi;
    int offset_j = blocksize * bj;
    int offset_r = blocksize * r;

    int i = threadIdx.y;
    int j = threadIdx.x;

    // DS[0:bibs-1][:] = B[bi][bj] = D[bibs:(bi+1)bs-1][bjbs:(bj+1)bs-1]
    // DS[bibs:2bibs-1][:] = B[bi][r] = D[bibs:(bi+1)bs-1][rbs:(r+1)bs-1]
    // DS[2bibs:3bibs-1][:] = B[r][bi] = D[rbs:(r+1)bs-1][bjbs:(bj+1)bs-1]
    DS[ij2ind(i, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_j, N)];
    DS[ij2ind(i+blocksize, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_r, N)];
    DS[ij2ind(i+2*blocksize, j, blocksize)] = D[ij2ind(i+offset_r, j+offset_j, N)];
    __syncthreads();

    // DS[i][j] = min{ DS[i][j], DS[i+bs][k] + DS[k+2bs][j] }
    for (int k = 0; k < blocksize; k++) {
        if (DS[ij2ind(i, j, blocksize)] > DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)]) {
            DS[ij2ind(i, j, blocksize)] = DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)];
            if (r == bi) DS[ij2ind(i+2*blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
            if (r == bj) DS[ij2ind(i+blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
        }
        __syncthreads();
    }

    // DS[i][j] = D[i+bsbi][j+bsbj]
    D[ij2ind(i+offset_i, j+offset_j, N)] = DS[ij2ind(i, j, blocksize)];
}

int main(int argc, char* argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, dev);

    int blockDimWidth = (int) sqrt(dp.maxThreadsPerBlock);

    if (argc < 3) {
        printf("not enough arguments.\n");
        return 0;
    }

    int blocksize;
    int MAX_BLOCKSIZE = blockDimWidth;

    if (argc >= 4) blocksize = atoi(argv[3]);
    else blocksize = blockDimWidth;

    if (blocksize > MAX_BLOCKSIZE) blocksize = MAX_BLOCKSIZE;

    // TODO: Read file and get meta data
    FILE *infile = fopen(argv[1], "r");
    int m;
    int N;
    fscanf(infile, "%d %d", &N, &m);

    if (blocksize > N) blocksize = N;
    if (blockDimWidth > blocksize) blockDimWidth = blocksize;
    int N_ext = N + (blocksize - ((N-1) % blocksize + 1));

    dim3 block(blockDimWidth, blockDimWidth);
    printf("Blocking factor: %d\n", blocksize);

    // TODO: Allocate memory (pinned/unpinned)
    hipHostMalloc((void**) &Dist, sizeof(int) * N_ext*N_ext, hipHostMallocDefault);
    hipMalloc((void**) &Dist_d, sizeof(int) * N_ext*N_ext);

    for (int i = 0; i < N_ext; ++i) {
        for (int j = 0; j < N_ext; ++j) {
            if (i == j) Dist[ij2ind(i, j, N_ext)] = 0;
            else        Dist[ij2ind(i, j, N_ext)] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        Dist[ij2ind(a, b, N_ext)] = v;
    }
    fclose(infile);

    // TODO: Copy values loaded from the file
    hipMemcpy((void*) Dist_d, (void*) Dist, sizeof(int) * N_ext*N_ext, hipMemcpyHostToDevice);

    // TODO: Updating list
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float phase1elapsed_millis = 0;
    float phase2elapsed_millis = 0;
    float phase3elapsed_millis = 0;
    float t;

    int num_blocks_per_column = (int) ceil((double) N_ext/blocksize);
    dim3 grid_1(2, num_blocks_per_column-1);
    dim3 grid_2(num_blocks_per_column-1, num_blocks_per_column-1);
    for (int r = 0; r < num_blocks_per_column; r++) {
        printf("\rCompute progress: %.2f%%", (float) r/num_blocks_per_column*100);

        hipEventRecord(start);
        updateList<<< 1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 0);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t, start, stop);
        phase1elapsed_millis += t;

        hipEventRecord(start);
        updateList<<< grid_1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 1);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t, start, stop);
        phase2elapsed_millis += t;

        hipEventRecord(start);
        updateList<<< grid_2, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t, start, stop);
        phase3elapsed_millis += t;

        printf("\rCompute progress: %.2f%%", (float) (r+1)/num_blocks_per_column*100);
    }
    printf("\n");

    // TODO: Copy final values
    hipMemcpy((void*) Dist, (void*) Dist_d, sizeof(int) * N_ext*N_ext, hipMemcpyDeviceToHost);

    // TODO: Write file
    if (OUTFILE == 1 && argc <= 4) {
        printf("Writing the file...\n");
        FILE *outfile = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (Dist[ij2ind(i, j, N_ext)] >= INF) fprintf(outfile, "INF ");
                else                                  fprintf(outfile, "%d ", Dist[ij2ind(i, j, N_ext)]);
            }
            fprintf(outfile, "\n");
        }
        fclose(outfile);
    }

    printf("phase_elapsed = (%.2f, %.2f, %.2f) ms\n", phase1elapsed_millis, phase2elapsed_millis, phase3elapsed_millis);

    // TODO: Free memory
    hipHostFree(Dist);
    hipFree(Dist_d);

    return 0;
}
