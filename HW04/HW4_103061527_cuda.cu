#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 500
#define INF (int) 1 << 29

int* Dist;
int* Dist_d;

__host__ __device__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

__global__ void updateList(int* list, int N, int k) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int i = idx / N;
    int j = idx % N;
    if (i < N && j < N && list[ij2ind(i, j, N)] > list[ij2ind(i, k, N)] + list[ij2ind(k, j, N)])
        list[ij2ind(i, j, N)] = list[ij2ind(i, k, N)] + list[ij2ind(k, j, N)];
}

int main(int argc, char* argv[]) {
    FILE *infile = fopen(argv[1], "r");
    int m;
    int N;
    fscanf(infile, "%d %d", &N, &m);

    // TODO: Allocate memory (pinned/unpinned)
    // Dist = (int*) malloc(sizeof(int) * N*N);
    hipHostMalloc((void**) &Dist, sizeof(int) * N*N, hipHostMallocDefault);
    hipMalloc((void**) &Dist_d, sizeof(int) * N*N);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (i == j) Dist[ij2ind(i, j, N)] = 0;
            else        Dist[ij2ind(i, j, N)] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        Dist[ij2ind(a, b, N)] = v;
    }

    // TODO: Copy values loaded from the file
    hipMemcpy((void*) Dist_d, (void*) Dist, sizeof(int) * N*N, hipMemcpyHostToDevice);

    // TODO: Updating list
    int num_blocks = N*N / NUM_THREADS + 1;
    for (int k = 0; k < N; k++) {
        updateList<<< num_blocks, NUM_THREADS >>>(Dist_d, N, k);
    }

    // TODO: Copy final values
    hipMemcpy((void*) Dist, (void*) Dist_d, sizeof(int) * N*N, hipMemcpyDeviceToHost);

    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (Dist[ij2ind(i, j, N)] >= INF) fprintf(outfile, "INF ");
            else                           fprintf(outfile, "%d ", Dist[ij2ind(i, j, N)]);
        }
        fprintf(outfile, "\n");
    }

    // TODO: Free memory
    hipHostFree(Dist);
    hipFree(Dist_d);

    return 0;
}
