#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#define OUTFILE 1
#define INF (int) 1 << 24

__device__ __host__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

int* Dist;
int* Dist_d;

__global__ void updateList(int* D, int blocksize, int N, int r, int blockDimWidth, int phase, int offset) {
    int bi, bj;
    switch(phase) {
        case 0:
            bi = r;
            bj = r;
            break;
        case 1:
            if (blockIdx.x == 1) {
                bj = r;
                bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            } else {
                bi = r;
                bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            }
            break;
        case 2:
            bi = blockIdx.x + offset;
            bj = blockIdx.y;
            break;
    }
    extern __shared__ int DS[];

    int offset_i = blocksize * bi;
    int offset_j = blocksize * bj;
    int offset_r = blocksize * r;

    // DS[0:bibs-1][:] = B[bi][bj] = D[bibs:(bi+1)bs-1][bjbs:(bj+1)bs-1]
    // DS[bibs:2bibs-1][:] = B[bi][r] = D[bibs:(bi+1)bs-1][rbs:(r+1)bs-1]
    // DS[2bibs:3bibs-1][:] = B[r][bi] = D[rbs:(r+1)bs-1][bjbs:(bj+1)bs-1]
    for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
            DS[ij2ind(i, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_j, N)];
            DS[ij2ind(i+blocksize, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_r, N)];
            DS[ij2ind(i+2*blocksize, j, blocksize)] = D[ij2ind(i+offset_r, j+offset_j, N)];
        }
    }
    __syncthreads();

    // DS[i][j] = min{ DS[i][j], DS[i+bs][k] + DS[k+2bs][j] }
    for (int k = 0; k < blocksize; k++) {
        for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
            for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
                if (DS[ij2ind(i, j, blocksize)] > DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)]) {
                    DS[ij2ind(i, j, blocksize)] = DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)];
                    if (r == bi) DS[ij2ind(i+2*blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                    if (r == bj) DS[ij2ind(i+blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                }
            }
        }
        __syncthreads();
    }

    for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
            // DS[i][j] = D[i+bsbi][j+bsbj]
            D[ij2ind(i+offset_i, j+offset_j, N)] = DS[ij2ind(i, j, blocksize)];
        }
    }
}

int main(int argc, char* argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, dev);

    int size, rank;

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    hipSetDevice(rank);
    int gpu_id = -1;
    hipGetDevice(&gpu_id);
    printf("Thread%d gets GPU%d\n", rank, gpu_id);

    int blockDimWidth = (int) sqrt(dp.maxThreadsPerBlock);

    if (argc < 3) {
        printf("not enough arguments.\n");
        return 0;
    }

    int blocksize;
    int MAX_BLOCKSIZE = (int) sqrt(dp.sharedMemPerBlock/3.0/sizeof(int));

    if (argc >= 4) blocksize = atoi(argv[3]);
    else blocksize = blockDimWidth;

    if (blocksize > MAX_BLOCKSIZE) blocksize = MAX_BLOCKSIZE;

    // TODO: Read file and get meta 
    FILE *infile;
    int m;
    int N;
    int N_ext;

    if (rank == 0) {
        infile = fopen(argv[1], "r");
        fscanf(infile, "%d %d", &N, &m);
        // Send N: MPI_Send(const void *buf, int count, MPI_Datatype datatype, int dest, int tag, MPI_Comm comm)
        MPI_Send(&N, 1, MPI_INT, 1, 0, MPI_COMM_WORLD);
    } else {
         // Recv N: MPI_Recv(void *buf, int count, MPI_Datatype datatype, int source, int tag, MPI_Comm comm, MPI_Status *status)
        MPI_Recv(&N, 1, MPI_INT, 0, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    if (blocksize > N) blocksize = N;
    if (blockDimWidth > blocksize) blockDimWidth = blocksize;
    N_ext = N + (blocksize - ((N-1) % blocksize + 1));

    dim3 block(blockDimWidth, blockDimWidth);
    if (rank == 0) printf("Blocking factor: %d\n", blocksize);

    // TODO: Allocate memory (pinned/unpinned)
    hipHostMalloc((void**) &Dist, sizeof(int) * N_ext*N_ext);
    hipMalloc((void**) &Dist_d, sizeof(int) * N_ext*N_ext);

    if (rank == 0) {
        for (int i = 0; i < N_ext; ++i) {
            for (int j = 0; j < N_ext; ++j) {
                if (i == j) Dist[ij2ind(i, j, N_ext)] = 0;
                else        Dist[ij2ind(i, j, N_ext)] = INF;
            }
        }

        while (--m >= 0) {
            int a, b, v;
            fscanf(infile, "%d %d %d", &a, &b, &v);
            --a, --b;
            Dist[ij2ind(a, b, N_ext)] = v;
        }
        fclose(infile);
        // Send Dist: MPI_Send(const void *buf, int count, MPI_Datatype datatype, int dest, int tag, MPI_Comm comm)
        MPI_Send(Dist, N_ext*N_ext, MPI_INT, 1, 0, MPI_COMM_WORLD);
    } else {
        // Recv Dist: MPI_Recv(void *buf, int count, MPI_Datatype datatype, int source, int tag, MPI_Comm comm, MPI_Status *status)
        MPI_Recv(Dist, N_ext*N_ext, MPI_INT, 0, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    // TODO: Updating list
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float phase1elapsed_millis = 0;
    float phase2elapsed_millis = 0;
    float phase3elapsed_millis = 0;
    float t;

    int num_blocks_per_column = (int) ceil((double) N_ext/blocksize);
    dim3 grid_1(2, num_blocks_per_column-1);

    int num_blocks_per_thread = num_blocks_per_column / size;
    int row_offset = num_blocks_per_thread * rank * blocksize;
    if (rank == size-1)
        num_blocks_per_thread += num_blocks_per_column % size;

    dim3 grid_2(num_blocks_per_thread, num_blocks_per_column);

    int cpy_idx = ij2ind(row_offset, 0, N_ext);
    hipMemcpy((void*) &Dist_d[cpy_idx], (void*) &Dist[cpy_idx], sizeof(int) * N_ext*blocksize*num_blocks_per_thread, hipMemcpyDeviceToHost);

    for (int r = 0; r < num_blocks_per_column; r++) {
        if (rank == 0) printf("\rCompute progress: %.2f%%", (float) r/num_blocks_per_column*100);

        int r_idx = ij2ind(r * blocksize, 0, N_ext);
        hipMemcpy((void*) &Dist_d[r_idx], (void*) &Dist[r_idx], sizeof(int) * N_ext * blocksize, hipMemcpyHostToDevice);

        if (rank == 0) hipEventRecord(start);
        updateList<<< 1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 0, -1);
        if (rank == 0) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);
            phase1elapsed_millis += t;
        }

        if (rank == 0) hipEventRecord(start);
        updateList<<< grid_1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 1, -1);
        if (rank == 0) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);
            phase2elapsed_millis += t;
        }

        if (rank == 0) hipEventRecord(start);

        updateList<<< grid_2, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_d, blocksize, N_ext, r, blockDimWidth, 2, row_offset/blocksize);

        if (rank == 0) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);
            phase3elapsed_millis += t;
        }

        hipMemcpy((void*) &Dist[cpy_idx], (void*) &Dist_d[cpy_idx], sizeof(int) * N_ext*blocksize*num_blocks_per_thread, hipMemcpyDeviceToHost);

        if (rank == 0) printf("\rCompute progress: %.2f%%", (float) (r+1)/num_blocks_per_column*100);

        // Exchange the results
        // MPI_Send(const void *buf, int count, MPI_Datatype datatype, int dest, int tag, MPI_Comm comm)
        // MPI_Recv(void *buf, int count, MPI_Datatype datatype, int source, int tag, MPI_Comm comm, MPI_Status *status)
        if (rank == 0) {
            int send_idx = 0;
            int send_cnt = N_ext*blocksize*num_blocks_per_thread;
            int recv_idx = ij2ind(num_blocks_per_thread * blocksize, 0, N_ext);
            int recv_cnt = N_ext*blocksize*(num_blocks_per_thread + num_blocks_per_column % size);
            MPI_Send(&Dist[send_idx], send_cnt, MPI_INT, 1, 0, MPI_COMM_WORLD);
            MPI_Recv(&Dist[recv_idx], recv_cnt, MPI_INT, 1, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        } else {
            int recv_idx = 0;
            int recv_cnt = N_ext*blocksize*(num_blocks_per_thread - num_blocks_per_column % size);
            int send_idx = ij2ind((num_blocks_per_thread - num_blocks_per_column % size) * blocksize, 0, N_ext);
            int send_cnt = N_ext*blocksize*num_blocks_per_thread;
            MPI_Recv(&Dist[recv_idx], recv_cnt, MPI_INT, 0, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            MPI_Send(&Dist[send_idx], send_cnt, MPI_INT, 0, 0, MPI_COMM_WORLD);
        }

        MPI_Barrier(MPI_COMM_WORLD);
    }
    printf("\n");

    // TODO: Write file
    if (OUTFILE == 1 && rank == 0 && argc <= 4) {
        printf("Writing the file...\n");
        FILE *outfile = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (Dist[ij2ind(i, j, N_ext)] >= INF) fprintf(outfile, "INF ");
                else                                  fprintf(outfile, "%d ", Dist[ij2ind(i, j, N_ext)]);
            }
            fprintf(outfile, "\n");
        }
        fclose(outfile);
    }

    printf("Process%d: phase_elapsed = (%.2f, %.2f, %.2f) ms\n", rank, phase1elapsed_millis, phase2elapsed_millis, phase3elapsed_millis);

    // TODO: Free memory
    hipHostFree(Dist);
    hipFree(Dist_d);

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();
    return 0;
}
