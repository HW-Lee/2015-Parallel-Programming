#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define OUTFILE 1
#define INF (int) 1 << 24

__device__ __host__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

int* Dist;
int** Dist_dt;

__global__ void updateList(int* D, int blocksize, int N, int r, int blockDimWidth, int phase, int rowIdx) {
    int bi, bj;
    switch(phase) {
        case 0:
            bi = r;
            bj = r;
            break;
        case 1:
            if (blockIdx.x == 1) {
                bj = r;
                bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            } else {
                bi = r;
                bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            }
            break;
        case 2:
            bi = (r + rowIdx + 1) % (int) ceil((double) N/blocksize);
            bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            break;
    }
    extern __shared__ int DS[];

    int offset_i = blocksize * bi;
    int offset_j = blocksize * bj;
    int offset_r = blocksize * r;

    // DS[0:bibs-1][:] = B[bi][bj] = D[bibs:(bi+1)bs-1][bjbs:(bj+1)bs-1]
    // DS[bibs:2bibs-1][:] = B[bi][r] = D[bibs:(bi+1)bs-1][rbs:(r+1)bs-1]
    // DS[2bibs:3bibs-1][:] = B[r][bi] = D[rbs:(r+1)bs-1][bjbs:(bj+1)bs-1]
    for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
            DS[ij2ind(i, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_j, N)];
            DS[ij2ind(i+blocksize, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_r, N)];
            DS[ij2ind(i+2*blocksize, j, blocksize)] = D[ij2ind(i+offset_r, j+offset_j, N)];
        }
    }
    __syncthreads();

    // DS[i][j] = min{ DS[i][j], DS[i+bs][k] + DS[k+2bs][j] }
    for (int k = 0; k < blocksize; k++) {
        for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
            for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
                if (DS[ij2ind(i, j, blocksize)] > DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)]) {
                    DS[ij2ind(i, j, blocksize)] = DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)];
                    if (r == bi) DS[ij2ind(i+2*blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                    if (r == bj) DS[ij2ind(i+blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
                }
            }
        }
        __syncthreads();
    }

    for (int i = threadIdx.y; i < blocksize; i+=blockDimWidth) {
        for (int j = threadIdx.x; j < blocksize; j+=blockDimWidth) {
            // DS[i][j] = D[i+bsbi][j+bsbj]
            D[ij2ind(i+offset_i, j+offset_j, N)] = DS[ij2ind(i, j, blocksize)];
        }
    }
}

int main(int argc, char* argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, dev);

    int num_dev = 1;
    hipGetDeviceCount(&num_dev);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        int gpuid = -1;
        hipGetDevice(&gpuid);
        printf("Thread%d gets GPU%d\n", omp_get_thread_num(), gpuid);
    }

    int blockDimWidth = (int) sqrt(dp.maxThreadsPerBlock);

    if (argc < 3) {
        printf("not enough arguments.\n");
        return 0;
    }

    int blocksize;
    int MAX_BLOCKSIZE = (int) sqrt(dp.sharedMemPerBlock/3.0/sizeof(int));
    if (argc >= 4) blocksize = atoi(argv[3]);
    else blocksize = blockDimWidth;

    if (blocksize > MAX_BLOCKSIZE) blocksize = MAX_BLOCKSIZE;
    if (blockDimWidth > blocksize) blockDimWidth = blocksize;

    // TODO: Read file and get meta data
    FILE *infile = fopen(argv[1], "r");
    int m;
    int N;
    fscanf(infile, "%d %d", &N, &m);

    if (blocksize > N) blocksize = N;
    int N_ext = N + (blocksize - ((N-1) % blocksize + 1));

    dim3 block(blockDimWidth, blockDimWidth);
    printf("Blocking factor: %d\n", blocksize);

    // TODO: Allocate memory (pinned/unpinned)
    hipHostMalloc((void**) &Dist, sizeof(int) * N_ext*N_ext, hipHostMallocDefault);
    Dist_dt = (int**) malloc(sizeof(int*) * num_dev);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        hipMalloc((void**) &Dist_dt[omp_get_thread_num()], sizeof(int) * N_ext*N_ext);
    }

    for (int i = 0; i < N_ext; ++i) {
        for (int j = 0; j < N_ext; ++j) {
            if (i == j) Dist[ij2ind(i, j, N_ext)] = 0;
            else        Dist[ij2ind(i, j, N_ext)] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        Dist[ij2ind(a, b, N_ext)] = v;
    }
    fclose(infile);

    // TODO: Updating list
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float phase1elapsed_millis = 0;
    float phase2elapsed_millis = 0;
    float phase3elapsed_millis = 0;
    float t;

    int num_blocks_per_column = (int) ceil((double) N_ext/blocksize);
    dim3 grid_1(2, num_blocks_per_column-1);
    dim3 grid_2(1, num_blocks_per_column-1);

    #pragma omp parallel num_threads(num_dev)
    {
        int t_id = omp_get_thread_num();
        hipSetDevice(t_id);

        for (int r = 0; r < num_blocks_per_column; r++) {
            if (t_id == 0) printf("\rCompute progress: %.2f%%", (float) r/num_blocks_per_column*100);

            hipMemcpy((void*) Dist_dt[t_id], (void*) Dist, sizeof(int) * N_ext*N_ext, hipMemcpyHostToDevice);

            if (t_id == 0) hipEventRecord(start);
            updateList<<< 1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 0, -1);
            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase1elapsed_millis += t;
            }

            if (t_id == 0) hipEventRecord(start);
            updateList<<< grid_1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 1, -1);
            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase2elapsed_millis += t;
            }

            if (t_id == 0) hipEventRecord(start);
            if (t_id == 0) hipMemcpy((void*) Dist, (void*) Dist_dt[0], sizeof(int) * N_ext*N_ext, hipMemcpyDeviceToHost);
            #pragma omp barrier

            hipMemcpy((void*) Dist_dt[t_id], (void*) Dist, sizeof(int) * N_ext*N_ext, hipMemcpyHostToDevice);
            #pragma omp for schedule(dynamic)
            for (int i = 0; i < num_blocks_per_column-1; i++) {
                updateList<<< grid_2, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 2, i);
                int idx = ij2ind( ((r+1+i) % num_blocks_per_column)*blocksize, 0, N_ext );
                hipMemcpy((void*) &(Dist[idx]), (void*) &(Dist_dt[t_id][idx]), sizeof(int)*N_ext*blocksize, hipMemcpyDeviceToHost);
            }
            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase3elapsed_millis += t;
            }

            if (t_id == 0) printf("\rCompute progress: %.2f%%", (float) (r+1)/num_blocks_per_column*100);

            #pragma omp barrier
        }
    }
    printf("\n");

    // TODO: Write file
    if (OUTFILE == 1) {
        printf("Writing the file...\n");
        FILE *outfile = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (Dist[ij2ind(i, j, N_ext)] >= INF) fprintf(outfile, "INF ");
                else                                  fprintf(outfile, "%d ", Dist[ij2ind(i, j, N_ext)]);
            }
            fprintf(outfile, "\n");
        }
        fclose(outfile);
    }

    printf("phase_elapsed = (%.2f, %.2f, %.2f) ms\n", phase1elapsed_millis, phase2elapsed_millis, phase3elapsed_millis);

    // TODO: Free memory
    hipHostFree(Dist);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        hipFree(Dist_dt[omp_get_thread_num()]);
    }

    return 0;
}