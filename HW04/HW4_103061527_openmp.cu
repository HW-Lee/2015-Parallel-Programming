#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define OUTFILE 1
#define INF (int) 1 << 24

__device__ __host__ int ij2ind(int i, int j, int N) {
    return i*N+j;
}

int* Dist;
int** Dist_dt;

__global__ void updateList(int* D, int blocksize, int N, int r, int blockDimWidth, int phase, int offset) {
    int bi, bj;
    switch(phase) {
        case 0:
            bi = r;
            bj = r;
            break;
        case 1:
            if (blockIdx.x == 1) {
                bj = r;
                bi = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            } else {
                bi = r;
                bj = (r + blockIdx.y + 1) % (int) ceil((double) N/blocksize);
            }
            break;
        case 2:
            bi = blockIdx.x + offset;
            bj = blockIdx.y;
            break;
    }
    extern __shared__ int DS[];

    int offset_i = blocksize * bi;
    int offset_j = blocksize * bj;
    int offset_r = blocksize * r;

    int i = threadIdx.y;
    int j = threadIdx.x;

    // DS[0:bibs-1][:] = B[bi][bj] = D[bibs:(bi+1)bs-1][bjbs:(bj+1)bs-1]
    // DS[bibs:2bibs-1][:] = B[bi][r] = D[bibs:(bi+1)bs-1][rbs:(r+1)bs-1]
    // DS[2bibs:3bibs-1][:] = B[r][bi] = D[rbs:(r+1)bs-1][bjbs:(bj+1)bs-1]
    DS[ij2ind(i, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_j, N)];
    DS[ij2ind(i+blocksize, j, blocksize)] = D[ij2ind(i+offset_i, j+offset_r, N)];
    DS[ij2ind(i+2*blocksize, j, blocksize)] = D[ij2ind(i+offset_r, j+offset_j, N)];
    __syncthreads();

    // DS[i][j] = min{ DS[i][j], DS[i+bs][k] + DS[k+2bs][j] }
    for (int k = 0; k < blocksize; k++) {
        if (DS[ij2ind(i, j, blocksize)] > DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)]) {
            DS[ij2ind(i, j, blocksize)] = DS[ij2ind(i+blocksize, k, blocksize)] + DS[ij2ind(k+2*blocksize, j, blocksize)];
            if (r == bi) DS[ij2ind(i+2*blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
            if (r == bj) DS[ij2ind(i+blocksize, j, blocksize)] = DS[ij2ind(i, j, blocksize)];
        }
        __syncthreads();
    }

    // DS[i][j] = D[i+bsbi][j+bsbj]
    D[ij2ind(i+offset_i, j+offset_j, N)] = DS[ij2ind(i, j, blocksize)];
}

int main(int argc, char* argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, dev);

    int num_dev = 1;
    hipGetDeviceCount(&num_dev);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        int gpuid = -1;
        hipGetDevice(&gpuid);
        printf("Thread%d gets GPU%d\n", omp_get_thread_num(), gpuid);
    }

    int blockDimWidth = (int) sqrt(dp.maxThreadsPerBlock);

    if (argc < 3) {
        printf("not enough arguments.\n");
        return 0;
    }

    int blocksize;
    int MAX_BLOCKSIZE = blockDimWidth;

    if (argc >= 4) blocksize = atoi(argv[3]);
    else blocksize = blockDimWidth;

    if (blocksize > MAX_BLOCKSIZE) blocksize = MAX_BLOCKSIZE;

    // TODO: Read file and get meta data
    FILE *infile = fopen(argv[1], "r");
    int m;
    int N;
    fscanf(infile, "%d %d", &N, &m);

    if (blocksize > N) blocksize = N;
    if (blockDimWidth > blocksize) blockDimWidth = blocksize;
    int N_ext = N + (blocksize - ((N-1) % blocksize + 1));

    dim3 block(blockDimWidth, blockDimWidth);
    printf("Blocking factor: %d\n", blocksize);

    // TODO: Allocate memory (pinned/unpinned)
    hipHostMalloc((void**) &Dist, sizeof(int) * N_ext*N_ext, hipHostMallocDefault);
    Dist_dt = (int**) malloc(sizeof(int*) * num_dev);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        hipMalloc((void**) &Dist_dt[omp_get_thread_num()], sizeof(int) * N_ext*N_ext);
    }

    for (int i = 0; i < N_ext; ++i) {
        for (int j = 0; j < N_ext; ++j) {
            if (i == j) Dist[ij2ind(i, j, N_ext)] = 0;
            else        Dist[ij2ind(i, j, N_ext)] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        Dist[ij2ind(a, b, N_ext)] = v;
    }
    fclose(infile);

    // TODO: Updating list
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float phase1elapsed_millis = 0;
    float phase2elapsed_millis = 0;
    float phase3elapsed_millis = 0;
    float t;

    int num_blocks_per_column = (int) ceil((double) N_ext/blocksize);
    dim3 grid_1(2, num_blocks_per_column-1);

    #pragma omp parallel num_threads(num_dev)
    {
        int t_id = omp_get_thread_num();
        hipSetDevice(t_id);

        int num_blocks_per_thread = num_blocks_per_column / num_dev;
        int row_offset = num_blocks_per_thread * t_id * blocksize;
        if (t_id == num_dev-1)
            num_blocks_per_thread += num_blocks_per_column % num_dev;

        dim3 grid_2(num_blocks_per_thread, num_blocks_per_column);

        int cpy_idx = ij2ind(row_offset, 0, N_ext);
        hipMemcpy((void*) &(Dist_dt[t_id][cpy_idx]), (void*) &(Dist[cpy_idx]), sizeof(int) * N_ext*blocksize*num_blocks_per_thread, hipMemcpyDeviceToHost);

        for (int r = 0; r < num_blocks_per_column; r++) {
            if (t_id == 0) printf("\rCompute progress: %.2f%%", (float) r/num_blocks_per_column*100);

            int r_idx = ij2ind(r * blocksize, 0, N_ext);

            if (r >= row_offset/blocksize && r < (row_offset/blocksize + num_blocks_per_thread)) {
                hipMemcpy((void*) &(Dist[r_idx]), (void*) &(Dist_dt[t_id][r_idx]), sizeof(int) * N_ext * blocksize, hipMemcpyDeviceToHost);
            }
            #pragma omp barrier

            hipMemcpy((void*) &(Dist_dt[t_id][r_idx]), (void*) &(Dist[r_idx]), sizeof(int) * N_ext * blocksize, hipMemcpyHostToDevice);

            if (t_id == 0) hipEventRecord(start);
            updateList<<< 1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 0, -1);
            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase1elapsed_millis += t;
            }

            if (t_id == 0) hipEventRecord(start);
            updateList<<< grid_1, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 1, -1);
            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase2elapsed_millis += t;
            }

            if (t_id == 0) hipEventRecord(start);

            updateList<<< grid_2, block, sizeof(int) * 3*blocksize*blocksize >>>(Dist_dt[t_id], blocksize, N_ext, r, blockDimWidth, 2, row_offset/blocksize);

            if (t_id == 0) {
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&t, start, stop);
                phase3elapsed_millis += t;
            }

            if (t_id == 0) printf("\rCompute progress: %.2f%%", (float) (r+1)/num_blocks_per_column*100);
        }

        hipMemcpy((void*) &(Dist[cpy_idx]), (void*) &(Dist_dt[t_id][cpy_idx]), sizeof(int) * N_ext*blocksize*num_blocks_per_thread, hipMemcpyDeviceToHost);
        #pragma omp barrier
    }
    printf("\n");

    // TODO: Write file
    if (OUTFILE == 1 && argc <= 4) {
        printf("Writing the file...\n");
        FILE *outfile = fopen(argv[2], "w");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (Dist[ij2ind(i, j, N_ext)] >= INF) fprintf(outfile, "INF ");
                else                                  fprintf(outfile, "%d ", Dist[ij2ind(i, j, N_ext)]);
            }
            fprintf(outfile, "\n");
        }
        fclose(outfile);
    }

    printf("phase_elapsed = (%.2f, %.2f, %.2f) ms\n", phase1elapsed_millis, phase2elapsed_millis, phase3elapsed_millis);

    // TODO: Free memory
    hipHostFree(Dist);
    #pragma omp parallel num_threads(num_dev)
    {
        hipSetDevice(omp_get_thread_num());
        hipFree(Dist_dt[omp_get_thread_num()]);
    }

    return 0;
}
