
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8
#define BLOCK_SIZE 1024
#define THREAD_SIZE 512

unsigned char *image_s = NULL;     // source image array
unsigned char *image_t = NULL;     // target image array
FILE *fp_s = NULL;                 // source file handler
FILE *fp_t = NULL;                 // target file handler

unsigned int   width, height;      // image width, image height
unsigned int   rgb_raw_data_offset;// RGB raw data offset
unsigned char  bit_per_pixel;      // bit per pixel
unsigned short byte_per_pixel;     // byte per pixel

// bitmap header
unsigned char header[54] = {
	0x42,        // identity : B
	0x4d,        // identity : M
	0, 0, 0, 0,  // file size
	0, 0,        // reserved1
	0, 0,        // reserved2
	54, 0, 0, 0, // RGB data offset
	40, 0, 0, 0, // struct BITMAPINFOHEADER size
	0, 0, 0, 0,  // bmp width
	0, 0, 0, 0,  // bmp height
	1, 0,        // planes
	24, 0,       // bit per pixel
	0, 0, 0, 0,  // compression
	0, 0, 0, 0,  // data size
	0, 0, 0, 0,  // h resolution
	0, 0, 0, 0,  // v resolution 
	0, 0, 0, 0,  // used colors
	0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
// Task 2: Put mask[][][] into Shared Memroy
int mask[MASK_N][MASK_X][MASK_Y] = {
	{{ -1, -4, -6, -4, -1},
	 { -2, -8,-12, -8, -2},
	 {  0,  0,  0,  0,  0},
	 {  2,  8, 12,  8,  2},
	 {  1,  4,  6,  4,  1}}
,
	{{ -1, -2,  0,  2,  1},
	 { -4, -8,  0,  8,  4},
	 { -6,-12,  0, 12,  6},
	 { -4, -8,  0,  8,  4},
	 { -1, -2,  0,  2,  1}}
};

int
read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET); 
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET); 
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// Task 3: Assign image_s to Pinnned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	// image_s = (unsigned char *) malloc((size_t)width * height * byte_per_pixel);
	hipHostMalloc( (void **) &image_s, (size_t) width * height * byte_per_pixel , hipHostMallocDefault);
	if (image_s == NULL) {
		printf("malloc images_s error\n");
		return -1;
	}

	// Task 3: Assign image_t to Pinned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	// image_t = (unsigned char *) malloc((size_t) width * height * byte_per_pixel);
	hipHostMalloc( (void **) &image_t, (size_t) width * height * byte_per_pixel , hipHostMallocDefault);
	if (image_t == NULL) {
		printf("malloc image_t error\n");
		return -1;
	}

	fread(image_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

int
sobel () {
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;

	// Task 2: Put mask[][][] into Shared Memory
	// Hint  : Please declare it in kernel function
	//         Then use some threads to move data from global memory to shared memory
	//         Remember to __syncthreads() after it's done <WHY?>

	// Task 1: Relabel x, y into combination of blockIdx, threadIdx ... etc
	// Hint A: We do not have enough threads for each pixels in the image, so what should we do?
	// Hint B: Maybe you can map each y to different threads in different blocks
	for (y = 0; y < height; ++y) {
		for (x = 0; x < width; ++x) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i*3+2] * val[i*3+2];
				totalG += val[i*3+1] * val[i*3+1];
				totalB += val[i*3+0] * val[i*3+0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			image_t[ byte_per_pixel * (width * y + x) + 2 ] = cR;
			image_t[ byte_per_pixel * (width * y + x) + 1 ] = cG;
			image_t[ byte_per_pixel * (width * y + x) + 0 ] = cB;
		}
	}

	return 0;
}

int
write_bmp (const char *fname_t) {
	unsigned int file_size; // file size

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size  
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(image_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int
init_device ()
{	// Task 1: Device (GPU) Initialization
	// Hint  : cudaSetDevice()
	hipSetDevice(1);
	return 0;
}

__global__  void sobel_Kernel(unsigned char* image_s, unsigned char* image_t, int width, int height, int byte_per_pixel, int* mask_arr) {
	// int mask_gpu[MASK_N][MASK_X][MASK_Y] = {
	// 	{{ -1, -4, -6, -4, -1},
	// 	 { -2, -8,-12, -8, -2},
	// 	 {  0,  0,  0,  0,  0},
	// 	 {  2,  8, 12,  8,  2},
	// 	 {  1,  4,  6,  4,  1}}
	// ,
	// 	{{ -1, -2,  0,  2,  1},
	// 	 { -4, -8,  0,  8,  4},
	// 	 { -6,-12,  0, 12,  6},
	// 	 { -4, -8,  0,  8,  4},
	// 	 { -1, -2,  0,  2,  1}}
	// };

	__shared__ int mask_gpu[MASK_N][MASK_X][MASK_Y];

	if ( threadIdx.x < MASK_N * MASK_X * MASK_Y ) {
		int n = threadIdx.x / (MASK_X * MASK_Y);
		int x = (threadIdx.x % (MASK_X * MASK_Y)) / MASK_Y;
		int y = (threadIdx.x % (MASK_X * MASK_Y)) % MASK_Y;
		mask_gpu[n][x][y] = mask_arr[threadIdx.x];
	}
	__syncthreads();

	int i, u, v;
	int R, G, B;
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;

	// (blockIdx.x, threadIdx.x)
	int x;
	int y;

	for (x = blockIdx.x; x < width; x += BLOCK_SIZE) {
		for (y = threadIdx.x; y < height; y += THREAD_SIZE) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask_gpu[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask_gpu[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask_gpu[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i*3+2] * val[i*3+2];
				totalG += val[i*3+1] * val[i*3+1];
				totalB += val[i*3+0] * val[i*3+0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			image_t[ byte_per_pixel * (width * y + x) + 2 ] = cR;
			image_t[ byte_per_pixel * (width * y + x) + 1 ] = cG;
			image_t[ byte_per_pixel * (width * y + x) + 0 ] = cB;
		}
	}
}

int
main(int argc, char **argv) {
	init_device();

	const char *input = "candy.bmp";
	if (argc > 1) input = argv[1];
	read_bmp(input); // 24 bit gray level image

	unsigned char *d_image_s, *d_image_t;
	int *mask_arr;

	// Task 1: Allocate memory on GPU
	// Hint  : cudaMalloc ()
	//         What do we need to store on GPU? (input image, output image, ...)
	hipMalloc( (void **) &d_image_s, (size_t) width * height * byte_per_pixel );
	hipMalloc( (void **) &d_image_t, (size_t) width * height * byte_per_pixel );
	hipMalloc( (void **) &mask_arr, MASK_N * MASK_X * MASK_Y * sizeof(int) );

	// Task 1: Memory copy from Host to Device (GPU)
	// Hint  : cudaMemcpy ( dst, src, count , cudaMemcpyHostToDevice )
	hipMemcpy( d_image_s, image_s, (size_t) width * height * byte_per_pixel, hipMemcpyHostToDevice );
	hipMemcpy( mask_arr, **mask, MASK_N * MASK_X * MASK_Y * sizeof(int), hipMemcpyHostToDevice );

	// Task 1: Modify sobel() to CUDA kernel function
	// Hint  : sobel_Kernel <<< ??? , ??? >>> ( ??? );
	sobel_Kernel<<< BLOCK_SIZE, THREAD_SIZE >>>( d_image_s, d_image_t, width, height, byte_per_pixel, mask_arr );

	// Task 1: Memory Copy from Device (GPU) to Host
	// Hint  : cudaMemcpy ( ... , cudaMemcpyDeviceToHost )
	hipMemcpy( image_t, d_image_t, (size_t) width * height * byte_per_pixel, hipMemcpyDeviceToHost );

	// Task 1: Free memory on device
	// Hint  : cudaFree ( ... )
	hipFree( d_image_s );
	hipFree( d_image_t );

	write_bmp("out.bmp");

	// Task 3: Free Pinned memory
	// Hint  : replace free ( ... ) by cudaFreeHost ( ... )
	// free (image_s);
	// free (image_t);
	hipHostFree( image_s );
	hipHostFree( image_t );
}
